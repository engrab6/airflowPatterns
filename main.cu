#include "hip/hip_runtime.h"
/*
Lattice Boltzmann method (LBM)
*/

#include<iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

typedef float real;
#define ROUND(x)( (x)-int(x) <= real(0.5) ? int(x) : (int(x)+1) )  //Define Round
#define makeEven(number) ((number%2==0)?(number):(number+1))  //Make Even Number


// problem parameters
/*
const int L = 256*2;  // lattices for X direction
const int M = 128*1;  // lattices for y direction
const int N = 128*1;  //lattices for z direction)
*/

       const int L=360;                           // lattices for X direction

      const int M_temp=ROUND((0.305/0.914)*L);  //0.457=width of the geometry in y directon
      const int M=makeEven(M_temp);             // lattices for y direction

      const int N_temp=ROUND((0.457/0.914)*L);  //0.305=height of the geometry in z directon
      const int N=makeEven(N_temp);                //lattices for z direction




const double inlet_dim=.101;           // Inlet size
const double outlet_dim=.101;          // Outlet Size

const double x_length=.914;           // Length of the geometry in x direction
const int outlet_diameter=ROUND((outlet_dim/x_length)*L);  //Nondimensionalization of outlet size
const int inlet_diameter=ROUND((inlet_dim/x_length)*L);  //Nondimensionalization of outlet size



const int TIME_STEPS = 2000;  // number of time steps for which the simulation is run
const int avg_time=500;
//const int NDIR = 19;           // number of discrete velocity directions used in the D2Q9 model
const double al = 0.2;
const double DENSITY = 1.0;          // fluid density in lattice units
const double u0 = .1;    // lid velocity in lattice units
const double re = 5000;  // Re =
const double nu=(u0*double(inlet_diameter))/re;
const double cs=0.1;
//const double nu =0.02;
const double pr=0.71;
const double ra=100000.0;
const double alpha=nu/pr;
//const double gbeta=ra*nu*alpha/(double(M*M*M));
//const double uref=sqrt(gbeta*double(N)/(ra*pr));
//const double uref=sqrt(gbeta*double(M));
//const double Ma=sqrt(3.0)*uref;
 const double tw=1.;
 const double tc=.0;
// const double u0 = 0.1;//uref;


// /*
      /*
      const double inlet_dim=.101;           // Inlet size
      const double outlet_dim=.101;          // Outlet Size

      const double x_length=.914;           // Length of the geometry in x direction
      const int outlet_diameter=ROUND((outlet_dim/x_length)*L);  //Nondimensionalization of outlet size
      const int inlet_diameter=ROUND((inlet_dim/x_length)*L);  //Nondimensionalization of outlet size

    */


//Partition 

		const double pw=4;  
		const int x1=((L+0)/2)-pw;                               //  starting coordinate of parttion along x direction
      		const int x2=((L+0)/2);                                  //  ending coordinate of parttion along x direction

    //--When inlet is in xy-plane --//
      const int inlet_size= inlet_diameter/2.0;
      // Here z=0 for all coordinate meaning the south/front wall 
      const int x3=(L+0)/4-inlet_size;             //  starting  coordinate of inlet along x direction
      const int y3=(M+0)/2-inlet_size;             //  starting  coordinate of inlet along y direction
      const int x4=(L+0)/4+inlet_size;             //  starting  coordinate of inlet along x direction
      const int y4=(M+0)/2+inlet_size;             //  starting  coordinate of inlet along y direction
       
          //--When inlet is in xy-plane --//
      const int outlet_size= outlet_diameter/2.0;
      // Here z=N for all coordinate meaning the north/back wall 
      const int x5=L/2+L/4-outlet_size;             //  starting  coordinate of outlet along x direction
      const int y5=(M+0)/2-outlet_size;             //  starting  coordinate of outlet along y direction
      const int x6=L/2+L/4+outlet_size;             //  starting  coordinate of outlet along x direction
      const int y6=(M+0)/2+outlet_size;             //  starting  coordinate of outlet along y direction

     //  */

      const int z2=N-1;                             //  starting coordinate of parttion along z direction
     /* const int z2=N/2;                           //  ending coordinate of parttion along z direction
      const int y4=0;
      const int y5=M-1; 
	*/

	const int y7=M/2 ;
        






#include "initial.h"
#include "kernel.h"
int main(int argc, char *argv[])
{
   //double *ux, *uy, rho for CPU;
hipSetDevice(1);
    double * ux = (double *) malloc (L*M*N*sizeof(double));
    double * uy = (double *) malloc (L*M*N*sizeof(double));
    double * uz = (double *) malloc (L*M*N*sizeof(double));
    double * rho = (double *) malloc (L*M*N*sizeof(double)); 
    double * mean_u = (double *) malloc (L*M*N*sizeof(double));
    double * mean_v = (double *) malloc (L*M*N*sizeof(double));
    double * mean_w = (double *) malloc (L*M*N*sizeof(double));

   //--Random Number Genaration--//
	 double * rands= (double *) malloc (L*M*N*sizeof(double));
        
	srand ( time ( NULL));
        int i;
        int s=L*M*N;
        for (i=0;i<s;i++){
        rands[i]=(double)rand()/RAND_MAX*2.0-1.0;
         }

     //--Random Number BlockEND--// 

    double * ups_h = (double *) malloc (L*M*N*sizeof(double));
    double * vps_h = (double *) malloc (L*M*N*sizeof(double));
    double * wps_h = (double *) malloc (L*M*N*sizeof(double));
    double * uvp_h = (double *) malloc (L*M*N*sizeof(double));
    double * vwp_h = (double *) malloc (L*M*N*sizeof(double));
    double * uwp_h = (double *) malloc (L*M*N*sizeof(double));
    double * t_h = (double *) malloc (L*M*N*sizeof(double));
    double * mean_t = (double *) malloc (L*M*N*sizeof(double));
    double * tps_h = (double *) malloc (L*M*N*sizeof(double));

    // allocate memory on the GPU for probability distribution function f_i
    double *d_f0, *d_f1, *d_f2, *d_f3, *d_f4, *d_f5, *d_f6, *d_f7,*d_f8,*d_f9,*d_f10, *d_f11, *d_f12, *d_f13, *d_f14, *d_f15, *d_f16, *d_f17,*d_f18;
    hipMalloc((void **)&d_f0,L*M*N*sizeof(double));
    hipMalloc((void **)&d_f1,L*M*N*sizeof(double));
    hipMalloc((void **)&d_f2,L*M*N*sizeof(double));
    hipMalloc((void **)&d_f3,L*M*N*sizeof(double));
    hipMalloc((void **)&d_f4,L*M*N*sizeof(double));
    hipMalloc((void **)&d_f5,L*M*N*sizeof(double));
    hipMalloc((void **)&d_f6,L*M*N*sizeof(double));
    hipMalloc((void **)&d_f7,L*M*N*sizeof(double));
    hipMalloc((void **)&d_f8,L*M*N*sizeof(double));
    hipMalloc((void **)&d_f9,L*M*N*sizeof(double));
    hipMalloc((void **)&d_f10,L*M*N*sizeof(double));
    hipMalloc((void **)&d_f11,L*M*N*sizeof(double));
    hipMalloc((void **)&d_f12,L*M*N*sizeof(double));
    hipMalloc((void **)&d_f13,L*M*N*sizeof(double));
    hipMalloc((void **)&d_f14,L*M*N*sizeof(double));
    hipMalloc((void **)&d_f15,L*M*N*sizeof(double));
    hipMalloc((void **)&d_f16,L*M*N*sizeof(double));
    hipMalloc((void **)&d_f17,L*M*N*sizeof(double));
    hipMalloc((void **)&d_f18,L*M*N*sizeof(double));

double *mf0, *mf1, *mf2, *mf3, *mf4, *mf5, *mf6, *mf7,*mf8,*mf9,*mf10, *mf11, *mf12, *mf13, *mf14, *mf15, *mf16, *mf17,*mf18;
    hipMalloc((void **)&mf0,L*M*N*sizeof(double));
    hipMalloc((void **)&mf1,L*M*N*sizeof(double));
    hipMalloc((void **)&mf2,L*M*N*sizeof(double));
    hipMalloc((void **)&mf3,L*M*N*sizeof(double));
    hipMalloc((void **)&mf4,L*M*N*sizeof(double));
    hipMalloc((void **)&mf5,L*M*N*sizeof(double));
    hipMalloc((void **)&mf6,L*M*N*sizeof(double));
    hipMalloc((void **)&mf7,L*M*N*sizeof(double));
    hipMalloc((void **)&mf8,L*M*N*sizeof(double));
    hipMalloc((void **)&mf9,L*M*N*sizeof(double));
    hipMalloc((void **)&mf10,L*M*N*sizeof(double));
    hipMalloc((void **)&mf11,L*M*N*sizeof(double));
    hipMalloc((void **)&mf12,L*M*N*sizeof(double));
    hipMalloc((void **)&mf13,L*M*N*sizeof(double));
    hipMalloc((void **)&mf14,L*M*N*sizeof(double));
    hipMalloc((void **)&mf15,L*M*N*sizeof(double));
    hipMalloc((void **)&mf16,L*M*N*sizeof(double));
    hipMalloc((void **)&mf17,L*M*N*sizeof(double));
    hipMalloc((void **)&mf18,L*M*N*sizeof(double));

   double  *d_g1, *d_g2, *d_g3, *d_g4, *d_g5,*d_g6;

        hipMalloc((void **)&d_g1,L*M*N*sizeof(double));
        hipMalloc((void **)&d_g2,L*M*N*sizeof(double));
        hipMalloc((void **)&d_g3,L*M*N*sizeof(double));
        hipMalloc((void **)&d_g4,L*M*N*sizeof(double));
        hipMalloc((void **)&d_g5,L*M*N*sizeof(double));
        hipMalloc((void **)&d_g6,L*M*N*sizeof(double));
double *gpost1,*gpost2,*gpost3,*gpost4,*gpost5,*gpost6;

     hipMalloc((void **)&gpost1,L*M*N*sizeof(double));
     hipMalloc((void **)&gpost2,L*M*N*sizeof(double));
     hipMalloc((void **)&gpost3,L*M*N*sizeof(double));
     hipMalloc((void **)&gpost4,L*M*N*sizeof(double));
     hipMalloc((void **)&gpost5,L*M*N*sizeof(double));
     hipMalloc((void **)&gpost6,L*M*N*sizeof(double));

// allocate memory on the GPU for post collision distribution function fpost_i

double *fpost0,*fpost1,*fpost2,*fpost3,*fpost4,*fpost5,*fpost6,*fpost7,*fpost8,*fpost9, *fpost10,*fpost11,*fpost12,*fpost13,*fpost14,*fpost15,*fpost16,
         *fpost17,*fpost18;
    hipMalloc((void **)&fpost0,L*M*N*sizeof(double));
    hipMalloc((void **)&fpost1,L*M*N*sizeof(double));
    hipMalloc((void **)&fpost2,L*M*N*sizeof(double));
    hipMalloc((void **)&fpost3,L*M*N*sizeof(double));
    hipMalloc((void **)&fpost4,L*M*N*sizeof(double));
    hipMalloc((void **)&fpost5,L*M*N*sizeof(double));
    hipMalloc((void **)&fpost6,L*M*N*sizeof(double));
    hipMalloc((void **)&fpost7,L*M*N*sizeof(double));
    hipMalloc((void **)&fpost8,L*M*N*sizeof(double));
    hipMalloc((void **)&fpost9,L*M*N*sizeof(double));
    hipMalloc((void **)&fpost10,L*M*N*sizeof(double));
    hipMalloc((void **)&fpost11,L*M*N*sizeof(double));
    hipMalloc((void **)&fpost12,L*M*N*sizeof(double));
    hipMalloc((void **)&fpost13,L*M*N*sizeof(double));
    hipMalloc((void **)&fpost14,L*M*N*sizeof(double));
    hipMalloc((void **)&fpost15,L*M*N*sizeof(double));
    hipMalloc((void **)&fpost16,L*M*N*sizeof(double));
    hipMalloc((void **)&fpost17,L*M*N*sizeof(double));
    hipMalloc((void **)&fpost18,L*M*N*sizeof(double));

// allocate memory on the GPU for velocity, density and relaxation parameter tau

    double *d_rho, *d_ux, *d_uy,*d_uz,*omega,*d_t,*rand_d;
    hipMalloc((void **)&d_rho,L*M*N*sizeof(double));
    hipMalloc((void **)&d_ux,L*M*N*sizeof(double));
    hipMalloc((void **)&d_uy,L*M*N*sizeof(double));
    hipMalloc((void **)&d_uz,L*M*N*sizeof(double));
    hipMalloc((void **)&omega,L*M*N*sizeof(double));
     hipMalloc((void **)&d_t,L*M*N*sizeof(double));
    hipMalloc((void **)&rand_d,L*M*N*sizeof(double));

  //  hipMalloc((void **)&omega,L*M*N*sizeof(double));
    double *umean ,*vmean,*wmean,*ups,*vps,*wps,*uvp,*vwp,*uwp,*tps,*tmean;
    hipMalloc((void **)&umean,L*M*N*sizeof(double));
    hipMalloc((void **)&vmean,L*M*N*sizeof(double));
    hipMalloc((void **)&wmean,L*M*N*sizeof(double));
    hipMalloc((void **)&ups,L*M*N*sizeof(double));
    hipMalloc((void **)&vps,L*M*N*sizeof(double));
    hipMalloc((void **)&wps,L*M*N*sizeof(double));
    hipMalloc((void **)&uvp,L*M*N*sizeof(double));
    hipMalloc((void **)&vwp,L*M*N*sizeof(double));
    hipMalloc((void **)&uwp,L*M*N*sizeof(double));
    hipMalloc((void **)&tmean,L*M*N*sizeof(double));
    hipMalloc((void **)&tps,L*M*N*sizeof(double));
    
    //Assigning value in the random variable array

     hipMemcpy(rand_d,rands,L*M*N * sizeof(double),hipMemcpyHostToDevice);

    // assign a 3D distribution of CUDA "threads" within each CUDA "block"
    dim3 grid_size(M,N,1);
    dim3 block_size(L,1,1);
   

initialize<<<grid_size, block_size>>>(d_rho, d_ux, d_uy,d_uz,d_t,
       d_f0, d_f1, d_f2, d_f3, d_f4, d_f5, d_f6, d_f7, d_f8,d_f9, d_f10, d_f11, d_f12, d_f13, d_f14, d_f15, d_f16, d_f17, d_f18,
       umean ,vmean,wmean,ups,vps,wps,uvp,vwp,uwp,tps,tmean,d_g1,d_g2,d_g3,d_g4,d_g5,d_g6,rand_d);
    // time integration
    int time=0;
   // int avg_step;
    while(time<TIME_STEPS) {

        time++;
       // avg_step=TIME_STEPS-time;
//std::cout<< time << "\n";
 kernel<<<grid_size, block_size>>>(d_rho, d_ux, d_uy,d_uz,d_t,
               d_f0, d_f1, d_f2, d_f3, d_f4, d_f5, d_f6, d_f7, d_f8,d_f9, d_f10, d_f11, d_f12, d_f13, d_f14, d_f15, d_f16, d_f17, d_f18,
         fpost0,fpost1,fpost2,fpost3,fpost4,fpost5,fpost6,fpost7,fpost8,fpost9,fpost10,fpost11,fpost12,fpost13,fpost14,fpost15,fpost16,fpost17,fpost18
      ,umean ,vmean,wmean,ups,vps,wps,uvp,vwp,uwp,tps,time,mf0, mf1, mf2, mf3, mf4, mf5, mf6, mf7, mf8,mf9, mf10, mf11, mf12, mf13, mf14, mf15, mf16, mf17, mf18,omega
      ,d_g1,d_g2,d_g3,d_g4,d_g5,d_g6, gpost1,gpost2, gpost3,gpost4,gpost5,gpost6,tmean,rand_d);


std::cout<<"Time step = " << time << "\n";
    }
    hipMemcpy(t_h,d_t,L*M*N*sizeof(double),hipMemcpyDeviceToHost);
// hipMemcpy(ux,d_ux,L*M*N*sizeof(double),hipMemcpyDeviceToHost);
// hipMemcpy(uy,d_uy,L*M*N*sizeof(double),hipMemcpyDeviceToHost);
// hipMemcpy(uz,d_uz,L*M*N*sizeof(double),hipMemcpyDeviceToHost); 
 //hipMemcpy(rho,d_rho,L*M*N*sizeof(double),hipMemcpyDeviceToHost);
 hipMemcpy(mean_u,umean,L*M*N*sizeof(double),hipMemcpyDeviceToHost);
 hipMemcpy(mean_v,vmean,L*M*N*sizeof(double),hipMemcpyDeviceToHost);
 hipMemcpy(mean_w,wmean,L*M*N*sizeof(double),hipMemcpyDeviceToHost);
 hipMemcpy(mean_t,tmean,L*M*N*sizeof(double),hipMemcpyDeviceToHost);

hipMemcpy(ups_h,ups,L*M*N*sizeof(double),hipMemcpyDeviceToHost);
hipMemcpy(vps_h,vps,L*M*N*sizeof(double),hipMemcpyDeviceToHost);
 hipMemcpy(wps_h,wps,L*M*N*sizeof(double),hipMemcpyDeviceToHost);
 hipMemcpy(uvp_h,uvp,L*M*N*sizeof(double),hipMemcpyDeviceToHost);
 hipMemcpy(vwp_h,vwp,L*M*N*sizeof(double),hipMemcpyDeviceToHost);
hipMemcpy(uwp_h,uwp,L*M*N*sizeof(double),hipMemcpyDeviceToHost);
hipMemcpy(tps_h,tps,L*M*N*sizeof(double),hipMemcpyDeviceToHost);
    
hipDeviceSynchronize();



FILE *fp,*fp1,*fp2,*fp3,*fp4,*fp5,*fp6,*fp7,*fp8,*fp9,*fp10,*fp11,*fp12,*fp13,*fp14,*fp15,*fp16,*fp17,*fp18,*fp19,*fp20,*fp21;


// Velocity profiles for different x positions ...

    fp1=fopen("Mean_vel_x_point1.dat","w");
    for(int j=0;j<=M-1;j++){
    fprintf(fp1,"%10.7f %16.14f %16.14f %16.14f",float(j)/float(M-1),mean_u[L/10+L*j+M*L*N/2]/(u0*(avg_time)),mean_v[L/10+L*j+M*L*N/2]/(u0*(avg_time)),
						mean_w[L/10+L*j+M*L*N/2]/(u0*(avg_time)));
    fprintf(fp1, "\n");
      }
    fclose(fp1);
    
 fp2=fopen("Mean_vel_x_point25.dat","w");
    for(int j=0;j<=M-1;j++){
    fprintf(fp2," %10.7f %16.14f %16.14f %16.14f",float(j)/float(M-1),mean_u[L/4+L*j+M*L*N/2]/(u0*(avg_time)),mean_v[L/4+L*j+M*L*N/2]/(u0*(avg_time)),
						mean_w[L/4+L*j+M*L*N/2]/(u0*(avg_time)));
    fprintf(fp2, "\n");
      }
    fclose(fp2);
 
 fp3=fopen("Mean_vel_x_point40.dat","w");
    for(int j=0;j<=M-1;j++){
    fprintf(fp3," %10.7f %16.14f %16.14f %16.14f",float(j)/float(M-1),mean_u[(2*L)/5+L*j+M*L*N/2]/(u0*(avg_time)),mean_v[(2*L)/5+L*j+M*L*N/2]/(u0*(avg_time)),
						mean_w[(2*L)/5+L*j+M*L*N/2]/(u0*(avg_time)));
    fprintf(fp3, "\n");
      }
    fclose(fp3);

 fp4=fopen("Mean_vel_x_point75.dat","w");
    for(int j=0;j<=M-1;j++){
    fprintf(fp4," %10.7f %16.14f %16.14f %16.14f",float(j)/float(M-1),mean_u[(3*L)/4+L*j+M*L*N/2]/(u0*(avg_time)),mean_v[(3*L)/4+L*j+M*L*N/2]/(u0*(avg_time)),mean_w[(3*L)/4+L*j+M*L*N/2]/(u0*(avg_time)));
    fprintf(fp4, "\n");
      }
    fclose(fp4);


fp5=fopen("Mean_vel_x_point90.dat","w");
    for(int j=0;j<=M-1;j++){
    fprintf(fp5," %10.7f %16.14f %16.14f %16.14f",float(j)/float(M-1),mean_u[(9*L)/10+L*j+M*L*N/2]/(u0*(avg_time)),mean_v[(9*L)/10+L*j+M*L*N/2]/(u0*(avg_time)),mean_w[(9*L)/10+L*j+M*L*N/2]/(u0*(avg_time)));
    fprintf(fp5, "\n");
      }
    fclose(fp5);




fp6=fopen("Mean_vel_y_point125.dat","w");
    for(int i=0;i<=L-1;i++){
    fprintf(fp6," %10.7f %16.14f %16.14f %16.14f",float(i)/float(L-1),mean_u[i+L*(4*L)/5+M*L*N/2]/(u0*(avg_time)),mean_v[i+L*(4*L)/5+M*L*N/2]/(u0*(avg_time)),mean_w[i+L*(4*L)/5+M*L*N/2]/(u0*(avg_time)));
    fprintf(fp6, "\n");
      }
    fclose(fp6);


fp7=fopen("Mean_vel_y_point25.dat","w");
    for(int i=0;i<=L-1;i++){
    fprintf(fp7," %10.7f %16.14f %16.14f %16.14f",float(i)/float(L-1),mean_u[i+L/4*L+M*L*N/2]/(u0*(avg_time)),mean_v[i+L/4*L+M*L*N/2]/(u0*(avg_time)),mean_w[i+L/4*L+M*L*N/2]/(u0*(avg_time)));
    fprintf(fp7, "\n");
      }
    fclose(fp7);



 fp=fopen("Mean_vel_x_point1.dat","w");
    for(int j=0;j<=M-1;j++){
    fprintf(fp,"%16.14f %16.14f %16.14f %10.7f ",mean_u[L/10+L*j+M*L*N/2]/(u0*(avg_time)),mean_v[L/10+L*j+M*L*N/2]/(u0*(avg_time)),
						mean_w[L/10+L*j+M*L*N/2]/(u0*(avg_time)), float(j)/float(M-1));
    fprintf(fp, "\n");
      }
    fclose(fp);
 fp=fopen("Mean_vel_x_point1.dat","w");
    for(int j=0;j<=M-1;j++){
    fprintf(fp,"%16.14f %16.14f %16.14f %10.7f ",mean_u[L/10+L*j+M*L*N/2]/(u0*(avg_time)),mean_v[L/10+L*j+M*L*N/2]/(u0*(avg_time)),
						mean_w[L/10+L*j+M*L*N/2]/(u0*(avg_time)), float(j)/float(M-1));
    fprintf(fp, "\n");
      }
    fclose(fp); fp=fopen("Mean_vel_x_point1.dat","w");
    for(int j=0;j<=M-1;j++){
    fprintf(fp,"%16.14f %16.14f %16.14f %10.7f ",mean_u[L/10+L*j+M*L*N/2]/(u0*(avg_time)),mean_v[L/10+L*j+M*L*N/2]/(u0*(avg_time)),
						mean_w[L/10+L*j+M*L*N/2]/(u0*(avg_time)), float(j)/float(M-1));
    fprintf(fp, "\n");
      }
    fclose(fp); fp=fopen("Mean_vel_x_point1.dat","w");
    for(int j=0;j<=M-1;j++){
    fprintf(fp,"%16.14f %16.14f %16.14f %10.7f ",mean_u[L/10+L*j+M*L*N/2]/(u0*(avg_time)),mean_v[L/10+L*j+M*L*N/2]/(u0*(avg_time)),
						mean_w[L/10+L*j+M*L*N/2]/(u0*(avg_time)), float(j)/float(M-1));
    fprintf(fp, "\n");
      }
    fclose(fp);	
  



// end of files

    fp=fopen("Mean_UV_vector.dat","w");
    fprintf(fp, "variable='x','y' ,'u','v' ""\n");
    fprintf(fp, "zone i=" "%5d",M);
    fprintf(fp, ", j=" "%5d",L);
    fprintf(fp,  ", f=point""\n");
    for(int i=0;i<=L-1;i++){
    for(int j=0;j<=M-1;j++){
//    fprintf(fp,"%f %f %15e %15e", float(i)/float(M-1), float(j)/float(M-1),ux[i+L*j+M*L*(N/2)]/u0,uy[i+L*j+M*L*(N/2)]/u0);
    fprintf(fp,"%5f %5f %15e %15e %15e", float(i)/float(M), float(j)/float(M),mean_u[i+L*j+M*L*(N/2)]/(u0*(avg_time)),mean_v[i+L*j+M*L*(N/2)]/(u0*(avg_time)),mean_w[i+L*j+M*L*(N/2)]/(u0*(avg_time)));
    fprintf(fp, "\n");
       }
      }
    fprintf(fp, "GEOMETRY M=GRID, FC=WHITE,F=POINT" "\n");
    fprintf(fp, "2" "\n");
    fprintf(fp, "4" "\n");
    fprintf(fp,"%f %f ",(x1/float(M/1)),0.0) ;
    fprintf(fp, "\n");
    fprintf(fp,"%f %f ",(x1/float(M/1)),(M/2)/float(M/1)) ;
    fprintf(fp, "\n");
    fprintf(fp,"%f %f ",(x2/float(M/1)),(M/2)/float(M/1)) ;
    fprintf(fp, "\n");
    fprintf(fp,"%f %f ",(x2/float(M/1)),0.0) ;
    fprintf(fp, "\n");
    fprintf(fp, "2" "\n");
    fprintf(fp,"%f %f ",(x1/float(M/1)),0.0) ;
    fprintf(fp, "\n");
    fprintf(fp,"%f %f ",(x2/float(M/1)),0.0) ;
    fprintf(fp, "\n");


    fclose(fp);


fp15=fopen("Mean_UW_vector_point50M.dat","w");
    fprintf(fp15, "variable='x','y' ,'u','v' ""\n");
    fprintf(fp15, "zone i=" "%5d",N);
    fprintf(fp15, ", j=" "%5d",L);
    fprintf(fp15,  ", f=point""\n");
    for(int i=0;i<=L-1;i++){
    for(int k=0;k<=N-1;k++){
//    fprintf(fp,"%f %f %15e %15e", float(i)/float(M-1), float(j)/float(M-1),ux[i+L*j+M*L*(N/2)]/u0,uy[i+L*j+M*L*(N/2)]/u0);
    fprintf(fp15,"%5f %5f %15e %15e %15e", float(i)/float(N), float(k)/float(N),mean_u[i+L*(M/2)+M*L*k]/(u0*(avg_time)),mean_v[i+L*(M/2)+M*L*k]/(u0*(avg_time)),mean_w[i+L*(M/2)+M*L*k]/(u0*(avg_time)));
    fprintf(fp15, "\n");
       }
      }
      fclose(fp15);



fp16=fopen("Mean_UW_vector_point25M.dat","w");
    fprintf(fp16, "variable='x','y' ,'u','v' ""\n");
    fprintf(fp16, "zone i=" "%5d",N);
    fprintf(fp16, ", j=" "%5d",L);
    fprintf(fp16,  ", f=point""\n");
    for(int i=0;i<=L-1;i++){
    for(int k=0;k<=N-1;k++){
//    fprintf(fp,"%f %f %15e %15e", float(i)/float(M-1), float(j)/float(M-1),ux[i+L*j+M*L*(N/2)]/u0,uy[i+L*j+M*L*(N/2)]/u0);
    fprintf(fp16,"%5f %5f %15e %15e %15e", float(i)/float(M), float(k)/float(N),mean_u[i+L*(M/4)+M*L*k]/(u0*(avg_time)),mean_v[i+L*(M/4)+M*L*k]/(u0*(avg_time)),mean_w[i+L*(M/4)+M*L*k]/(u0*(avg_time)));
    fprintf(fp16, "\n");
       }
      }
      fclose(fp16);



fp17=fopen("Mean_UW_vector_point75M.dat","w");
    fprintf(fp17, "variable='x','y' ,'u','v' ""\n");
    fprintf(fp17, "zone i=" "%5d",N);
    fprintf(fp17, ", j=" "%5d",L);
    fprintf(fp17,  ", f=point""\n");
    for(int i=0;i<=L-1;i++){
    for(int k=0;k<=N-1;k++){
//    fprintf(fp,"%f %f %15e %15e", float(i)/float(M-1), float(j)/float(M-1),ux[i+L*j+M*L*(N/2)]/u0,uy[i+L*j+M*L*(N/2)]/u0);
    fprintf(fp17,"%5f %5f %15e %15e %15e", float(i)/float(M), float(k)/float(N),mean_u[i+L*(3*M/4)+M*L*k]/(u0*(avg_time)),mean_v[i+L*(3*M/4)+M*L*k]/(u0*(avg_time)),mean_w[i+L*(3*M/4)+M*L*k]/(u0*(avg_time)));
    fprintf(fp17, "\n");
       }
      }
      fclose(fp17);

 fp8=fopen("isotherms_str_point50.dat","w");
    fprintf(fp8, "variable='x','y' ,'u','v' ""\n");
    fprintf(fp8, "zone i=" "%5d",M);
    fprintf(fp8, ", j=" "%5d",N);
    fprintf(fp8,  ", f=point""\n");
    for(int j=0;j<=M-1;j++){
    for(int k=0;k<=N-1;k++){
    fprintf(fp8,"%5f %5f %15e", float(j)/float(M-1), float(k)/float(N-1),mean_t[(L/2)+L*j+M*L*k]/(u0*(avg_time)));
    fprintf(fp8, "\n");
       }
      }
    fclose(fp8);


 fp18=fopen("isotherms_str_point25.dat","w");
    fprintf(fp18, "variable='x','y' ,'u','v' ""\n");
    fprintf(fp18, "zone i=" "%5d",M);
    fprintf(fp18, ", j=" "%5d",N);
    fprintf(fp18,  ", f=point""\n");
    for(int j=0;j<=M-1;j++){
    for(int k=0;k<=N-1;k++){
    fprintf(fp18,"%5f %5f %15e", float(j)/float(M-1), float(k)/float(N-1),mean_t[(L/4)+L*j+M*L*k]/(u0*(avg_time)));
    fprintf(fp18, "\n");
       }
      }
    fclose(fp18);

 fp19=fopen("isotherms_str_point75.dat","w");
    fprintf(fp19, "variable='x','y' ,'u','v' ""\n");
    fprintf(fp19, "zone i=" "%5d",M);
    fprintf(fp19, ", j=" "%5d",N);
    fprintf(fp19,  ", f=point""\n");
    for(int j=0;j<=M-1;j++){
    for(int k=0;k<=N-1;k++){
    fprintf(fp19,"%5f %5f %15e", float(j)/float(M-1), float(k)/float(N-1),mean_t[(3*L/4)+L*j+M*L*k]/(u0*(avg_time)));
    fprintf(fp19, "\n");
       }
      }
    fclose(fp19);





  /* fp1=fopen("Mean_WV_vector.dat","w");
    fprintf(fp1, "variable='x','y' ,'u','v' ""\n");
    fprintf(fp1, "zone i=" "%5d",M);
    fprintf(fp1, ", j=" "%5d",N);
    fprintf(fp1,  ", f=point""\n");
    for(int k=0;k<=N-1;k++){
    for(int j=0;j<=M-1;j++){    
    fprintf(fp1,"%f %f %15e %15e", float(k)/float(M-1), float(j)/float(M-1),mean_w[L/4+L*j+M*L*k]/(u0*(avg_time)),mean_v[L/4+L*j+M*L*k]/(u0*(avg_time)));
//    fprintf(fp1,"%f %f %15e %15e", float(j)/float(N-1), float(k)/float(N-1),vps_h[L/2+L*j+M*L*k]/(u0*avg_time),uvp_h[L/2+L*j+M*L*k]/(u0*avg_time));
//    fprintf(fp,"%f %f %15e %15e", float(i)/float(N-1), float(j)/float(N-1),ux[i+L*j+M*L*(N/2)]/u0,u0);
    fprintf(fp1, "\n");
       }
      }
    fclose(fp1);
    fp101=fopen("Mean_WU_vector.dat","w");
    fprintf(fp101, "variable='x','y' ,'u','v' ""\n");
    fprintf(fp101, "zone i=" "%5d",L);
    fprintf(fp101, ", j=" "%5d",N);
    fprintf(fp101,  ", f=point""\n");
    for(int i=0;i<=L-1;i++){
    for(int k=0;k<=N-1;k++){
    fprintf(fp101,"%f %f %15e %15e", float(k)/float(M-1), float(i)/float(M-1),mean_w[i+L*M/2+M*L*k]/(u0*(avg_time)),mean_u[i+L*M/2+M*L*k]/(u0*(avg_time)));
    fprintf(fp101, "\n");
     }
    }
   fclose(fp101);

    FILE *fp2,*fp3;  
    fp2=fopen("Mean_vel_u_midx.dat","w");
    for(int j=0;j<=M-1;j++){
    fprintf(fp2,"%16.14f %10.7f ",mean_u[L/2+L*j+M*L*N/2]/(u0*(avg_time)), float(j)/float(M-1));
    fprintf(fp2, "\n");
      }
    fclose(fp2);
    
    fp3=fopen("Mean_vel_v_midy.dat","w");
    for(int i=0;i<=L-1;i++){
    fprintf(fp3,"%16.14f  %10.7f ", float(i)/float(M-1),mean_v[i+L*M/2+M*L*N/2]/(u0*(avg_time)));
    fprintf(fp3, "\n");
      }
    fclose(fp3);
*/
    FILE *fp04,*fp05;
    fp04=fopen("rms_uvw_quarter_xy.dat","w");
    for(int k=1;k<=N-1;k++){
   fprintf(fp04,"%5f %15e %15e %15e", float(k)/float(N-1),sqrt(ups_h[L/4+L*M/2+M*L*k])/(u0*(avg_time)),sqrt(vps_h[L/4+L*M/2+M*L*k])/(u0*(avg_time)),sqrt(wps_h[L/4+L*M/2+M*L*k])/(u0*(avg_time))); 
    fprintf(fp04, "\n");
      }
    fclose(fp04);
    fp05=fopen("rms_uvw_quarter_yz.dat","w");
    for(int i=1;i<=L-1;i++){
   fprintf(fp05,"%5f %15e %15e %15e", float(i)/float(N-1),sqrt(ups_h[i+L*M/2+M*L*N/4])/(u0*(avg_time)), sqrt(vps_h[i+L*M/2+M*L*N/4])/(u0*(avg_time)),sqrt(wps_h[i+L*M/2+M*L*N/4])/(u0*(avg_time)));
   fprintf(fp05, "\n");
      }
    fclose(fp05);


   FILE *fp06,*fp07;
    fp06=fopen("temperature_tps_midx.dat","w");
    for(int k=1;k<=N-1;k++){
    fprintf(fp06,"%5f %15e ",float(k)/float(N-1),tps_h[L/2+L*M/2+M*L*k]/(u0*(avg_time)));
    fprintf(fp06, "\n");
      }
    fclose(fp06);


/*
    FILE *fp06,*fp07;
    fp06=fopen("Reynolds_uv_midx.dat","w");
    for(int k=1;k<=N-1;k++){
    fprintf(fp06,"%5f %15e ",float(k)/float(N-1),uwp_h[L/2+L*M/2+M*L*k]/(u0*u0*(avg_time)));
    fprintf(fp06, "\n");
      }
    fclose(fp06);
    fp07=fopen("Reynolds_uv_midy.dat","w");
    for(int i=1;i<L-1;i++){
   fprintf(fp07,"%5f %15e ", float(i)/float(N-1),uwp_h[i+L*M/2+M*L*N/2]/(u0*u0*(avg_time)));
    fprintf(fp07, "\n");
      }
    fclose(fp07);
*/

FILE *fp03,*fp002;
double snul=0.0;
double snur=0.0;
double snur1=0.0;
for(int j=0;j<=M-1;j++){
     for(int k=0;k<=N-1;k++){
         double rnul=(t_h[0+L*j+L*M*k]-t_h[1+L*j+L*M*k])*double(M);
         //double rnur=(t[(M-2)+M*j]-t[(M-1)+j*M])*double(M-1);
         float rnur=-0.5*(4.0*t_h[1+j*L+L*M*k]-3.0*t_h[0+L*j+L*M*k]-t_h[2+L*j+M*N*k])*double(M);
         //fprintf(fp002,"%d %f %f ", , rnul,rnur);
         //fprintf(fp002, "\n");
         snul=snul+rnul;
         snur=snur+rnur;
     }
}
for(int j=0;j<=M-1;j++){
    double rnul1=(t_h[0+L*j+L*M*N/2]-t_h[1+L*j+L*M*N/2])*double(M);
    //double rnur=(t[(M-2)+M*j]-t[(M-1)+j*M])*double(M-1);
    //float rnur=-0.5*(4.0*theta[(x2+1)+j*M]-3.0*theta[x2+M*j]-theta[(x2+2)+M*j])*float(N);
    //fprintf(fp002,"%d %f %f ", , rnul,rnur);
    //fprintf(fp002, "\n");
    snur1=snur1+rnul1;
    //snur=snur+rnur;
}

fp03=fopen("Avg_Nu_left.dat","w");
 //fprintf(fp0,"%f %f %f %f", ra, snul/double(x2-x1),snur/double(x2-x1),((snul)/1.0)/double(x2-1-x1));
  fprintf(fp03,"%f %f %f %f ", ra,1*(snul/double((M)*(M))), 1*(snur/double((M)*(M))),1*(snur1/double(M)));
   fprintf(fp03, "\n");
    fclose(fp03);
 FILE *fp005;
  fp005=fopen("Mean_Inst_temp.dat","w");
  fprintf(fp005, "variable='x','y' ,'u','v' ""\n");
  fprintf(fp005, "zone i=" "%5d",M);
  fprintf(fp005, ", j=" "%5d",L);
  fprintf(fp005,  ", f=point""\n");
  for(int i=0;i<=L-1;i++){
  for(int j=0;j<=M-1;j++){
  fprintf(fp005,"%8f %8f %15e %15e ", float(i)/float(M), float(j)/float(M),mean_t[i+L*j+M*L*N/2]/(avg_time),t_h[i+L*j+M*L*N/2]);
  fprintf(fp005, "\n");
  }
  }
 fclose(fp005);


 FILE *fp102;
  fp102=fopen("Mean_WV_vector_point25.dat","w");
  fprintf(fp102, "variable='x','y' ,'u','v' ""\n");
  fprintf(fp102, "zone i=" "%5d",N);
  fprintf(fp102, ", j=" "%5d",M);
  fprintf(fp102,  ", f=point""\n");
  for(int k=0;k<=N-1;k++){
  for(int j=0;j<=M-1;j++){
  fprintf(fp102,"%8f %8f %15e %15e", float(k)/float(N), float(j)/float(M),mean_v[(L/4)+L*j+M*L*k]/(u0*(avg_time)),mean_w[(L/4)+L*j+M*L*k]/(u0*(avg_time)));
  fprintf(fp102, "\n");
  }
  }
 fclose(fp102);
 
  fp14=fopen("Mean_WV_vector_point50.dat","w");
  fprintf(fp14, "variable='x','y' ,'u','v' ""\n");
  fprintf(fp14, "zone i=" "%5d",N);
  fprintf(fp14, ", j=" "%5d",M);
  fprintf(fp14,  ", f=point""\n");
  for(int k=0;k<=N-1;k++){
  for(int j=0;j<=M-1;j++){
  fprintf(fp14,"%8f %8f %15e %15e", float(k)/float(N), float(j)/float(M), mean_v[(L/2)+L*j+M*L*k]/(u0*(avg_time)),mean_w[(L/2)+L*j+M*L*k]/(u0*(avg_time)));
  fprintf(fp14, "\n");
  }
  }
 fclose(fp14);


  fp20=fopen("Mean_WV_vector_point75.dat","w");
  fprintf(fp20, "variable='x','y' ,'u','v' ""\n");
  fprintf(fp20, "zone i=" "%5d",N);
  fprintf(fp20, ", j=" "%5d",M);
  fprintf(fp20,  ", f=point""\n");
  for(int k=0;k<=N-1;k++){
  for(int j=0;j<=M-1;j++){
  fprintf(fp20,"%8f %8f %15e %15e", float(k)/float(N), float(j)/float(M), mean_v[(3*L/4)+L*j+M*L*k]/(u0*(avg_time)),mean_w[(3*L/4)+L*j+M*L*k]/(u0*(avg_time)));
  fprintf(fp20, "\n");
  }
  }
 fclose(fp20);


fp21=fopen("3DContour_temp.dat","w");
    fprintf(fp21, "variable='x','z' ,'y','T' " "\n");
    fprintf(fp21, "zone i=" "%5d",M);
    fprintf(fp21, ", k=" "%5d",N);
    fprintf(fp21, ", j=" "%5d",L);
    fprintf(fp21,  ", f=point""\n");
    for(int i=0;i<=L-1;i++){
    for(int j=0;j<=M-1;j++){
     for(int k=0;k<=N-1;k++){
      fprintf(fp21,"%f %f %f %15e", float(i)/float(M),float(j)/float(M), float(k)/float(M),t_h[i+L*j+M*L*k]);
 
 
	fprintf(fp21, "\n");
        }
    	}
	}


        fprintf(fp21,"GEOMETRY  M=GRID, FC=BLUE, T=LINE3D, F=POINT" "\n");
        fprintf(fp21,"5" "\n");
	
	 fprintf(fp21,"5" "\n");
         fprintf(fp21,"%f %f %f ",(x1/float(M/1)),0.0,0.0) ;
         fprintf(fp21, "\n");
        fprintf(fp21,"%f %f %f ",(x2/float(M/1)),0.0,0.0) ;
         fprintf(fp21, "\n");
         fprintf(fp21,"%f %f %f ",(x2/float(M/1)),y7/float(M/1),0.0) ;
         fprintf(fp21, "\n");
         fprintf(fp21,"%f %f %f ",(x1/float(M/1)),y7/float(M/1),0.0) ;
         fprintf(fp21, "\n");
	fprintf(fp21,"%f %f %f ",(x1/float(M/1)),0.0,0.0) ;
         fprintf(fp21, "\n");


        fprintf(fp21,"4" "\n");
         fprintf(fp21,"%f %f %f ",(x1/float(M/1)),0.0,0.0) ;
         fprintf(fp21, "\n");
         fprintf(fp21,"%f %f %f ",(x1/float(M/1)),0.0,z2/float(M/1)) ;
         fprintf(fp21, "\n");
        fprintf(fp21,"%f %f %f ",(x1/float(M/1)),y7/float(M/1),z2/float(M/1)) ;
         fprintf(fp21, "\n");
        fprintf(fp21,"%f %f %f ",(x1/float(M/1)),y7/float(M/1),0.0) ;
        fprintf(fp21, "\n");
        
         fprintf(fp21,"4" "\n");


        fprintf(fp21,"%f %f %f ",(x2/float(M/1)),y7/float(M/1),0.0) ;
        fprintf(fp21, "\n");
        fprintf(fp21,"%f %f %f ",(x2/float(M/1)),y7/float(M/1),z2/float(M/1)) ;
        fprintf(fp21, "\n");
        fprintf(fp21,"%f %f %f ",(x2/float(M/1)),0.0,z2/float(M/1)) ;
        fprintf(fp21, "\n");

        fprintf(fp21,"%f %f %f ",(x2/float(M/1)),0.0,0.0) ;
        fprintf(fp21, "\n");

        fprintf(fp21,"2" "\n");
        fprintf(fp21,"%f %f %f ",(x1/float(M/1)),y7/float(M/1),z2/float(M/1)) ;
        fprintf(fp21, "\n");
        fprintf(fp21,"%f %f %f ",(x2/float(M/1)),y7/float(M/1),z2/float(M/1)) ;
        fprintf(fp21, "\n");

        fprintf(fp21,"2" "\n");
        fprintf(fp21,"%f %f %f ",(x1/float(M/1)),0.0,z2/float(N/1)) ;
        fprintf(fp21, "\n");
        fprintf(fp21,"%f %f %f ",(x2/float(M/1)),0.0,z2/float(N/1)) ;
        fprintf(fp21, "\n");
fclose(fp21);




FILE *fp22;

fp22=fopen("3DContour_velocity.dat","w");
    fprintf(fp22, "variable='x','z' ,'y','V' " "\n");
    fprintf(fp22, "zone i=" "%5d",M);
    fprintf(fp22, ", k=" "%5d",N);
    fprintf(fp22, ", j=" "%5d",L);
    fprintf(fp22,  ", f=point""\n");
    for(int i=0;i<=L-1;i++){
    for(int j=0;j<=M-1;j++){
     for(int k=0;k<=N-1;k++){
      fprintf(fp22,"%f %f %f %15e", float(i)/float(M),float(j)/float(M), float(k)/float(M),mean_w[i+L*j+M*L*k]/(u0*(avg_time)));
        fprintf(fp22, "\n");
        }
        }
        }


        fprintf(fp22,"GEOMETRY  M=GRID, FC=BLUE, T=LINE3D, F=POINT" "\n");
       fprintf(fp22,"5" "\n");

         fprintf(fp22,"5" "\n");
         fprintf(fp22,"%f %f %f ",(x1/float(M/1)),0.0,0.0) ;
         fprintf(fp22, "\n");
        fprintf(fp22,"%f %f %f ",(x2/float(M/1)),0.0,0.0) ;
         fprintf(fp22, "\n");
         fprintf(fp22,"%f %f %f ",(x2/float(M/1)),y7/float(M/1),0.0) ;
         fprintf(fp22, "\n");
         fprintf(fp22,"%f %f %f ",(x1/float(M/1)),y7/float(M/1),0.0) ;
         fprintf(fp22, "\n");
        fprintf(fp22,"%f %f %f ",(x1/float(M/1)),0.0,0.0) ;
         fprintf(fp22, "\n");

 

         fprintf(fp22,"4" "\n");
         fprintf(fp22,"%f %f %f ",(x1/float(M/1)),0.0,0.0) ;
         fprintf(fp22, "\n");
         fprintf(fp22,"%f %f %f ",(x1/float(M/1)),0.0,z2/float(M/1)) ;
         fprintf(fp22, "\n");
        fprintf(fp22,"%f %f %f ",(x1/float(M/1)),y7/float(M/1),z2/float(M/1)) ;
         fprintf(fp22, "\n");
        fprintf(fp22,"%f %f %f ",(x1/float(M/1)),y7/float(M/1),0.0) ;
        fprintf(fp22, "\n");

        fprintf(fp22,"4" "\n");
	fprintf(fp22,"%f %f %f ",(x2/float(M/1)),y7/float(M/1),0.0) ;
        fprintf(fp22, "\n");
        fprintf(fp22,"%f %f %f ",(x2/float(M/1)),y7/float(M/1),z2/float(M/1)) ;
        fprintf(fp22, "\n");
        fprintf(fp22,"%f %f %f ",(x2/float(M/1)),0.0,z2/float(M/1)) ;
        fprintf(fp22, "\n");
        fprintf(fp22,"%f %f %f ",(x2/float(M/1)),0.0,0.0) ;

	fprintf(fp22,"2" "\n");
        fprintf(fp22,"%f %f %f ",(x1/float(M/1)),y7/float(M/1),z2/float(M/1)) ;
        fprintf(fp22, "\n");
        fprintf(fp22,"%f %f %f ",(x2/float(M/1)),y7/float(M/1),z2/float(M/1)) ;
        fprintf(fp22, "\n");

        fprintf(fp22,"2" "\n");
        fprintf(fp22,"%f %f %f ",(x1/float(M/1)),0.0,z2/float(N/1)) ;
        fprintf(fp22, "\n");
        fprintf(fp22,"%f %f %f ",(x2/float(M/1)),0.0,z2/float(N/1)) ;
        fprintf(fp22, "\n");




fclose(fp22);

    return 0;
}
